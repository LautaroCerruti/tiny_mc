#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include "params.h"

// CUDA kernel: one thread simulates one photon
__global__ void photon_kernel(float* heats, float* heats_squared, unsigned int photons_per_thread) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= PHOTONS/photons_per_thread) return;

    // Initialize per-thread RNG
    hiprandState state;
    //hiprand_init(seed, idx, 0, &state);
    hiprand_init(clock64(), idx, 0, &state);

    // Precompute constants
    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4f / MICRONS_PER_SHELL / (MU_A + MU_S);

    // Photon state
    float x = 0.0f, y = 0.0f, z = 0.0f;
    float u = 0.0f, v = 0.0f, w = 1.0f;
    float weight = 1.0f;

    unsigned int remaining_photons = photons_per_thread;

    // Photon life loop
    while (remaining_photons) {
        // Sample step length
        float rnd = hiprand_uniform(&state);
        float t = -logf(rnd);
        x += t * u;
        y += t * v;
        z += t * w;

        // Determine shell
        int shell = (int)(sqrtf(x*x + y*y + z*z) * shells_per_mfp);
        if (shell >= SHELLS) shell = SHELLS - 1;

        // Deposit energy
        float deposit = (1.0f - albedo) * weight;
        atomicAdd(&heats[shell], deposit);
        atomicAdd(&heats_squared[shell], deposit * deposit);

        // Update weight
        weight *= albedo;

        // Roulette for low-weight photons
        if (weight < 0.001f) {
            if (hiprand_uniform(&state) > 0.1f) {
                // Photon is absorbed
                remaining_photons--;
                x = 0.0f;
                y = 0.0f; 
                z = 0.0f;
                u = 0.0f;
                v = 0.0f;
                w = 1.0f;
                weight = 1.0f; // Reset weight for next photon
                continue; // Skip to next photon
            } else {
                weight *= 10f;
            }
        }

        // Scatter: sample new direction using rejection method
        float xi1, xi2, s;
        do {
            xi1 = 2.0f * hiprand_uniform(&state) - 1.0f;
            xi2 = 2.0f * hiprand_uniform(&state) - 1.0f;
            s = xi1*xi1 + xi2*xi2;
        } while (s > 1.0f);

        u = 2.0f * s - 1.0f;
        //float factor = sqrtf((1.0f - u*u) / s);
        float factor = 2.0f * sqrtf(1.0f - s);
        v = xi1 * factor;
        w = xi2 * factor;
    }
}

void launch_simulation(float *h_heats, float *h_heats_sq) {
    // Allocate and zero device arrays
    float *d_heats, *d_heats_sq;
    hipMalloc(&d_heats,  SHELLS * sizeof(float));
    hipMalloc(&d_heats_sq, SHELLS * sizeof(float));
    hipMemset(d_heats,  0, SHELLS * sizeof(float));
    hipMemset(d_heats_sq, 0, SHELLS * sizeof(float));

    // Choose launch parameters
    int threads = 256;
    int blocks = (PHOTONS + threads - 1) / threads;

    // Launch kernel with a time- or user-defined seed
    photon_kernel<<<blocks, threads>>>(d_heats, d_heats_sq, 12345ULL);
    hipDeviceSynchronize();

    // Copy results back to host...
    // float h_heats[SHELLS], h_heats_sq[SHELLS];
    // hipMemcpy(h_heats,    d_heats,    SHELLS * sizeof(float), hipMemcpyDeviceToHost);
    // hipMemcpy(h_heats_sq, d_heats_sq, SHELLS * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_heats);
    hipFree(d_heats_sq);
}
