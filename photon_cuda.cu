#include "hip/hip_runtime.h"
#include "photon_cuda.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <math.h>
#include "params.h"
#include "hip/hip_runtime_api.h"

// CUDA kernel: each thread simulates `photons_per_thread` photons
__global__ void photon_kernel_v1(float* heats, float* heats_squared, unsigned int photons_per_thread) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = (PHOTONS + photons_per_thread - 1) / photons_per_thread;
    if (idx >= total_threads) return;

    // Initialize per-thread RNG
    hiprandState state;
    //hiprand_init(seed, idx, 0, &state);
    hiprand_init(clock64(), idx, 0, &state);

    // Precompute constants
    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4f / MICRONS_PER_SHELL / (MU_A + MU_S);

    // Photon state
    float x = 0.0f, y = 0.0f, z = 0.0f;
    float u = 0.0f, v = 0.0f, w = 1.0f;
    float weight = 1.0f;

    unsigned int remaining_photons = photons_per_thread;

    // Photon life loop
    while (remaining_photons) {
        // Sample step length
        float rnd = hiprand_uniform(&state);
        float t = -logf(rnd);
        x += t * u;
        y += t * v;
        z += t * w;

        // Determine shell
        int shell = (int)(sqrtf(x*x + y*y + z*z) * shells_per_mfp);
        if (shell >= SHELLS) shell = SHELLS - 1;

        // Deposit energy
        float deposit = (1.0f - albedo) * weight;
        atomicAdd(&heats[shell], deposit);
        atomicAdd(&heats_squared[shell], deposit * deposit);

        // Update weight
        weight *= albedo;

        // Roulette for low-weight photons
        if (weight < 0.001f) {
            if (hiprand_uniform(&state) > 0.1f) {
                // Photon is absorbed
                remaining_photons--;
                x = 0.0f;
                y = 0.0f;
                z = 0.0f;
                u = 0.0f;
                v = 0.0f;
                w = 1.0f;
                weight = 1.0f;
                continue;
            } else {
                weight *= 10.0f;
            }
        }

        // Scatter: sample new direction using rejection method
        float xi1, xi2, s;
        do {
            xi1 = 2.0f * hiprand_uniform(&state) - 1.0f;
            xi2 = 2.0f * hiprand_uniform(&state) - 1.0f;
            s = xi1*xi1 + xi2*xi2;
        } while (s > 1.0f);

        u = 2.0f * s - 1.0f;
        //float factor = sqrtf((1.0f - u*u) / s);
        float factor = 2.0f * sqrtf(1.0f - s);
        v = xi1 * factor;
        w = xi2 * factor;
    }
}

void launch_simulation(float *h_heats, float *h_heats_sq, double *elapsed_time) {
    float *d_heats, *d_heats_sq;
    checkCudaCall(hipMalloc(&d_heats,  SHELLS * sizeof(float)));
    checkCudaCall(hipMalloc(&d_heats_sq, SHELLS * sizeof(float)));
    checkCudaCall(hipMemset(d_heats,  0, SHELLS * sizeof(float)));
    checkCudaCall(hipMemset(d_heats_sq, 0, SHELLS * sizeof(float)));

    hipEvent_t start, stop;
    checkCudaCall(hipEventCreate(&start));
    checkCudaCall(hipEventCreate(&stop));

    int blocks = (PHOTONS / GPU_PHOTONS_PER_THREAD) / GPU_THREADS;

    checkCudaCall(hipEventRecord(start, 0));

    photon_kernel_v1<<<blocks, GPU_THREADS>>>(d_heats, d_heats_sq, GPU_PHOTONS_PER_THREAD);

    checkCudaCall(hipEventRecord(stop, 0));
    checkCudaCall(hipEventSynchronize(stop));

    float ms = 0.0f;
    checkCudaCall(hipEventElapsedTime(&ms, start, stop));

    *elapsed_time = ms / 1000.0;

    checkCudaCall(hipMemcpy(h_heats, d_heats, SHELLS * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaCall(hipMemcpy(h_heats_sq, d_heats_sq, SHELLS * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipEventDestroy(start));
    checkCudaCall(hipEventDestroy(stop));
    checkCudaCall(hipFree(d_heats));
    checkCudaCall(hipFree(d_heats_sq));
}
