#include "hip/hip_runtime.h"
#include "photon_cuda.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <math.h>
#include "params.h"
#include "hip/hip_runtime_api.h"
#include <hip/hip_math_constants.h>
#include <cstdint>

struct Xoshiro128pState {
    uint32_t s0, s1, s2, s3;
};

__device__ __forceinline__ uint32_t rotl32(uint32_t x, int k) {
    return (x << k) | (x >> (32 - k));
}

__device__ __forceinline__ uint64_t splitmix64(uint64_t &state) {
    uint64_t z = (state += 0x9E3779B97F4A7C15ULL);
    z = (z ^ (z >> 30)) * 0xBF58476D1CE4E5B9ULL;
    z = (z ^ (z >> 27)) * 0x94D049BB133111EBULL;
    return z ^ (z >> 31);
}

__device__ __forceinline__ void xoshiro128p_init(uint64_t seed, Xoshiro128pState *st) {
    uint64_t sm = seed;
    st->s0 = (uint32_t)splitmix64(sm);
    st->s1 = (uint32_t)splitmix64(sm);
    st->s2 = (uint32_t)splitmix64(sm);
    st->s3 = (uint32_t)splitmix64(sm);
}

__device__ __forceinline__ float4 xoshiro128p_next4(Xoshiro128pState *st) {
    const float norm = 2.3283064365386963e-10f; // 1/2^32

    float4 r;
    uint32_t result, t;

    result = st->s0 + st->s3;
    r.x    = result * norm;
    t      = st->s1 << 9;
    st->s2 ^= st->s0;  st->s3 ^= st->s1;
    st->s1 ^= st->s2;  st->s0 ^= st->s3;
    st->s2 ^= t;       st->s3 = rotl32(st->s3, 11);

    result = st->s0 + st->s3;
    r.y    = result * norm;
    t      = st->s1 << 9;
    st->s2 ^= st->s0;  st->s3 ^= st->s1;
    st->s1 ^= st->s2;  st->s0 ^= st->s3;
    st->s2 ^= t;       st->s3 = rotl32(st->s3, 11);

    result = st->s0 + st->s3;
    r.z    = result * norm;
    t      = st->s1 << 9;
    st->s2 ^= st->s0;  st->s3 ^= st->s1;
    st->s1 ^= st->s2;  st->s0 ^= st->s3;
    st->s2 ^= t;       st->s3 = rotl32(st->s3, 11);

    result = st->s0 + st->s3;
    r.w    = result * norm;
    t      = st->s1 << 9;
    st->s2 ^= st->s0;  st->s3 ^= st->s1;
    st->s1 ^= st->s2;  st->s0 ^= st->s3;
    st->s2 ^= t;       st->s3 = rotl32(st->s3, 11);

    return r;
}

__device__ __forceinline__ float xoshiro128p_next(Xoshiro128pState *st) {
    uint32_t result = st->s0 + st->s3;
    uint32_t t      = st->s1 << 9;
    st->s2 ^= st->s0;  st->s3 ^= st->s1;
    st->s1 ^= st->s2;  st->s0 ^= st->s3;
    st->s2 ^= t;       st->s3 = rotl32(st->s3, 11);
    return result * (1.0f / 4294967296.0f);
}

__global__ void photon_kernel(float* heats, float* heats_squared, unsigned int photons_per_thread) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int lid  = tid%warpSize;

    __shared__ float s_heats[SHELLS][32];
    __shared__ float s_heats_sq[SHELLS][32];

    if (GPU_THREADS < SHELLS) {
        if (tid < warpSize) {
            for (int i = 0; i < SHELLS; i++) {
                s_heats[i][lid]    = 0.0f;
                s_heats_sq[i][lid] = 0.0f;
            }
        }
    } else {
        if (tid < SHELLS) {
            for (int j = 0; j < warpSize; j++) {
                s_heats[tid][j]    = 0.0f;
                s_heats_sq[tid][j] = 0.0f;
            }
        }
    }

    __syncthreads();

    Xoshiro128pState state;
    uint64_t seed = clock64() + (uint64_t)idx;
    xoshiro128p_init(seed, &state);

    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4f / MICRONS_PER_SHELL / (MU_A + MU_S);

    float x = 0.0f, y = 0.0f, z = 0.0f;
    float u = 0.0f, v = 0.0f, w = 1.0f;
    float weight = 1.0f;

    unsigned int remaining_photons = photons_per_thread;

    while (remaining_photons) {
        float t = -logf(xoshiro128p_next(&state));
        x += t * u;
        y += t * v;
        z += t * w;

        int shell = min(int(sqrtf(x*x + y*y + z*z) * shells_per_mfp), SHELLS-1);

        float deposit = (1.0f - albedo) * weight;
        atomicAdd(&s_heats[shell][lid], deposit);
        atomicAdd(&s_heats_sq[shell][lid], deposit*deposit);

        weight *= albedo;

        u = 2.0f * xoshiro128p_next(&state) - 1.0f;
        float temp = sqrtf(1.0f - u*u);
        float sin, cos;
        sincosf(2.0f * HIP_PI_F * xoshiro128p_next(&state), &sin, &cos);
        v = sin * temp;
        w = cos * temp;

        if (weight < 0.001f) {
            weight *= 10.0f;
            if (xoshiro128p_next(&state) > 0.1f) {
                x = 0.0f;
                y = 0.0f;
                z = 0.0f;
                u = 0.0f;
                v = 0.0f;
                w = 1.0f;
                weight = 1.0f;
                remaining_photons--;
            }
        }
    }

    __syncthreads();

    if (tid < warpSize) {
        for (int i = 0; i < SHELLS; i++) {
            float warp_heat = s_heats[i][lid];
            #define FULL_MASK 0xffffffff
            warp_heat += __shfl_down_sync(FULL_MASK, warp_heat, 16);
            warp_heat += __shfl_down_sync(FULL_MASK, warp_heat, 8);
            warp_heat += __shfl_down_sync(FULL_MASK, warp_heat, 4);
            warp_heat += __shfl_down_sync(FULL_MASK, warp_heat, 2);
            warp_heat += __shfl_down_sync(FULL_MASK, warp_heat, 1);
            if (lid == 0) {
                atomicAdd(&heats[i], warp_heat);
            }

            float warp_heat_sq = s_heats_sq[i][lid];
            warp_heat_sq += __shfl_down_sync(FULL_MASK, warp_heat_sq, 16);
            warp_heat_sq += __shfl_down_sync(FULL_MASK, warp_heat_sq, 8);
            warp_heat_sq += __shfl_down_sync(FULL_MASK, warp_heat_sq, 4);
            warp_heat_sq += __shfl_down_sync(FULL_MASK, warp_heat_sq, 2);
            warp_heat_sq += __shfl_down_sync(FULL_MASK, warp_heat_sq, 1);
            if (lid == 0) {
                atomicAdd(&heats_squared[i], warp_heat_sq);
            }
        }
    }
}

void launch_simulation(float *h_heats, float *h_heats_sq, double *elapsed_time) {
    float *d_heats, *d_heats_sq;
    checkCudaCall(hipMalloc(&d_heats,  SHELLS * sizeof(float)));
    checkCudaCall(hipMalloc(&d_heats_sq, SHELLS * sizeof(float)));
    checkCudaCall(hipMemset(d_heats,  0, SHELLS * sizeof(float)));
    checkCudaCall(hipMemset(d_heats_sq, 0, SHELLS * sizeof(float)));

    hipEvent_t start, stop;
    checkCudaCall(hipEventCreate(&start));
    checkCudaCall(hipEventCreate(&stop));

    unsigned int photons_per_thread = PHOTONS / (GPU_BLOCKS * GPU_THREADS);
    printf("# Launching %d blocks with %d threads each and %d photons per thread\n", GPU_BLOCKS, GPU_THREADS, photons_per_thread);

    checkCudaCall(hipEventRecord(start, 0));

    photon_kernel<<<GPU_BLOCKS, GPU_THREADS>>>(d_heats, d_heats_sq, photons_per_thread);

    checkCudaCall(hipEventRecord(stop, 0));
    checkCudaCall(hipEventSynchronize(stop));

    float ms = 0.0f;
    checkCudaCall(hipEventElapsedTime(&ms, start, stop));

    *elapsed_time = ms / 1000.0;

    checkCudaCall(hipMemcpy(h_heats, d_heats, SHELLS * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaCall(hipMemcpy(h_heats_sq, d_heats_sq, SHELLS * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipEventDestroy(start));
    checkCudaCall(hipEventDestroy(stop));
    checkCudaCall(hipFree(d_heats));
    checkCudaCall(hipFree(d_heats_sq));
}
