#include "hip/hip_runtime.h"
#include "photon_cuda.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <math.h>
#include "params.h"
#include "hip/hip_runtime_api.h"
#include <cstdint>

__global__ void photon_kernel_atomic_global(float* heats, float* heats_squared, unsigned int photons_per_thread) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize per-thread RNG
    hiprandState state;
    //hiprand_init(seed, idx, 0, &state);
    hiprand_init(clock64(), idx, 0, &state);

    // Precompute constants
    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4f / MICRONS_PER_SHELL / (MU_A + MU_S);

    // Photon state
    float x = 0.0f, y = 0.0f, z = 0.0f;
    float u = 0.0f, v = 0.0f, w = 1.0f;
    float weight = 1.0f;

    unsigned int remaining_photons = photons_per_thread;

    // Photon life loop
    while (remaining_photons) {
        // Sample step length
        float rnd = hiprand_uniform(&state);
        float t = -logf(rnd);
        x += t * u;
        y += t * v;
        z += t * w;

        // Determine shell
        int shell = (int)(sqrtf(x*x + y*y + z*z) * shells_per_mfp);
        if (shell >= SHELLS) shell = SHELLS - 1;

        // Deposit energy
        float deposit = (1.0f - albedo) * weight;
        atomicAdd(&heats[shell], deposit);
        atomicAdd(&heats_squared[shell], deposit * deposit);

        // Update weight
        weight *= albedo;

        // Roulette for low-weight photons
        if (weight < 0.001f) {
            if (hiprand_uniform(&state) > 0.1f) {
                // Photon is absorbed
                remaining_photons--;
                x = 0.0f;
                y = 0.0f;
                z = 0.0f;
                u = 0.0f;
                v = 0.0f;
                w = 1.0f;
                weight = 1.0f;
                continue;
            } else {
                weight *= 10.0f;
            }
        }

        // Scatter: sample new direction using rejection method
        float xi1, xi2, s;
        do {
            xi1 = 2.0f * hiprand_uniform(&state) - 1.0f;
            xi2 = 2.0f * hiprand_uniform(&state) - 1.0f;
            s = xi1*xi1 + xi2*xi2;
        } while (s > 1.0f);

        u = 2.0f * s - 1.0f;
        //float factor = sqrtf((1.0f - u*u) / s);
        float factor = 2.0f * sqrtf(1.0f - s);
        v = xi1 * factor;
        w = xi2 * factor;
    }
}

__global__ void photon_kernel_shared(float* heats, float* heats_squared, unsigned int photons_per_thread) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // apuntadores en shared memory
    __shared__ float s_heats[SHELLS];
    __shared__ float s_heats_sq[SHELLS];

    // Inicialización de shared (por los primeros SHELLS hilos)
    if (tid < SHELLS) {
        s_heats[tid]    = 0.0f;
        s_heats_sq[tid] = 0.0f;
    }
    
    __syncthreads();

    // Initialize per-thread RNG
    hiprandState state;
    //hiprand_init(seed, idx, 0, &state);
    hiprand_init(clock64(), idx, 0, &state);

    // Precompute constants
    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4f / MICRONS_PER_SHELL / (MU_A + MU_S);

    // Photon state
    float x = 0.0f, y = 0.0f, z = 0.0f;
    float u = 0.0f, v = 0.0f, w = 1.0f;
    float weight = 1.0f;

    unsigned int remaining_photons = photons_per_thread;

    // Photon life loop
    while (remaining_photons) {
        // Sample step length
        float rnd = hiprand_uniform(&state);
        float t = -logf(rnd);
        x += t * u;
        y += t * v;
        z += t * w;

        // Determine shell
        int shell = (int)(sqrtf(x*x + y*y + z*z) * shells_per_mfp);
        if (shell >= SHELLS) shell = SHELLS - 1;

        // Deposit energy
        float deposit = (1.0f - albedo) * weight;
        atomicAdd(&s_heats[shell], deposit);
        atomicAdd(&s_heats_sq[shell], deposit*deposit);

        // Update weight
        weight *= albedo;

        // Roulette for low-weight photons
        if (weight < 0.001f) {
            if (hiprand_uniform(&state) > 0.1f) {
                // Photon is absorbed
                remaining_photons--;
                x = 0.0f;
                y = 0.0f;
                z = 0.0f;
                u = 0.0f;
                v = 0.0f;
                w = 1.0f;
                weight = 1.0f;
                continue;
            } else {
                weight *= 10.0f;
            }
        }

        // Scatter: sample new direction using rejection method
        float xi1, xi2, s;
        do {
            xi1 = 2.0f * hiprand_uniform(&state) - 1.0f;
            xi2 = 2.0f * hiprand_uniform(&state) - 1.0f;
            s = xi1*xi1 + xi2*xi2;
        } while (s > 1.0f);

        u = 2.0f * s - 1.0f;
        //float factor = sqrtf((1.0f - u*u) / s);
        float factor = 2.0f * sqrtf(1.0f - s);
        v = xi1 * factor;
        w = xi2 * factor;
    }

    __syncthreads();

    if (tid < SHELLS) {
        atomicAdd(&heats[tid], s_heats[tid]);
        atomicAdd(&heats_squared[tid], s_heats_sq[tid]);
    }
}

__global__ void photon_kernel_polares(float* heats, float* heats_squared, unsigned int photons_per_thread) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // apuntadores en shared memory
    __shared__ float s_heats[SHELLS];
    __shared__ float s_heats_sq[SHELLS];

    // Inicialización de shared (por los primeros SHELLS hilos)
    if (tid < SHELLS) {
        s_heats[tid]    = 0.0f;
        s_heats_sq[tid] = 0.0f;
    }
    
    __syncthreads();

    // Initialize per-thread RNG
    hiprandState state;
    //hiprand_init(seed, idx, 0, &state);
    hiprand_init(clock64(), idx, 0, &state);

    // Precompute constants
    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4f / MICRONS_PER_SHELL / (MU_A + MU_S);

    // Photon state
    float x = 0.0f, y = 0.0f, z = 0.0f;
    float u = 0.0f, v = 0.0f, w = 1.0f;
    float weight = 1.0f;

    unsigned int remaining_photons = photons_per_thread;

    // Photon life loop
    while (remaining_photons) {
        // Sample step length
        float rnd = hiprand_uniform(&state);
        float t = -logf(rnd);
        x += t * u;
        y += t * v;
        z += t * w;

        // Determine shell
        int shell = (int)(sqrtf(x*x + y*y + z*z) * shells_per_mfp);
        if (shell >= SHELLS) shell = SHELLS - 1;

        // Deposit energy
        float deposit = (1.0f - albedo) * weight;
        atomicAdd(&s_heats[shell], deposit);
        atomicAdd(&s_heats_sq[shell], deposit*deposit);

        // Update weight
        weight *= albedo;

        // Roulette for low-weight photons
        if (weight < 0.001f) {
            if (hiprand_uniform(&state) > 0.1f) {
                // Photon is absorbed
                remaining_photons--;
                x = 0.0f;
                y = 0.0f;
                z = 0.0f;
                u = 0.0f;
                v = 0.0f;
                w = 1.0f;
                weight = 1.0f;
                continue;
            } else {
                weight *= 10.0f;
            }
        }

        float xi1, xi2, s, r, sin, cos;

        s = hiprand_uniform(&state);
        r = sqrtf(s);
        sincospif(2.0f * hiprand_uniform(&state), &sin, &cos);
        xi1 = r * cos;
        xi2 = r * sin;

        u = 2.0f * s - 1.0f;
        float factor = 2.0f * sqrtf(1.0f - s);
        v = xi1 * factor;
        w = xi2 * factor;
    }

    __syncthreads();

    if (tid < SHELLS) {
        atomicAdd(&heats[tid], s_heats[tid]);
        atomicAdd(&heats_squared[tid], s_heats_sq[tid]);
    }
}

struct Xoshiro128pState {
    uint32_t s0, s1, s2, s3;
};

__device__ __forceinline__ uint32_t rotl32(uint32_t x, int k) {
    return (x << k) | (x >> (32 - k));
}

__device__ __forceinline__ uint64_t splitmix64(uint64_t &state) {
    uint64_t z = (state += 0x9E3779B97F4A7C15ULL);
    z = (z ^ (z >> 30)) * 0xBF58476D1CE4E5B9ULL;
    z = (z ^ (z >> 27)) * 0x94D049BB133111EBULL;
    return z ^ (z >> 31);
}

__device__ __forceinline__ void xoshiro128p_init(uint64_t seed, Xoshiro128pState *st) {
    // Usamos SplitMix64 para generar cuatro palabras de 32 bits
    uint64_t sm = seed;
    st->s0 = (uint32_t)splitmix64(sm);
    st->s1 = (uint32_t)splitmix64(sm);
    st->s2 = (uint32_t)splitmix64(sm);
    st->s3 = (uint32_t)splitmix64(sm);
}

__device__ __forceinline__ float4 xoshiro128p_next4(Xoshiro128pState *st) {
    const float norm = 2.3283064365386963e-10f; // 1/2^32

    float4 r;
    uint32_t result, t;

    // — primera muestra —
    result = st->s0 + st->s3;
    r.x    = result * norm;
    t      = st->s1 << 9;
    st->s2 ^= st->s0;  st->s3 ^= st->s1;
    st->s1 ^= st->s2;  st->s0 ^= st->s3;
    st->s2 ^= t;       st->s3 = rotl32(st->s3, 11);

    // — segunda muestra —
    result = st->s0 + st->s3;
    r.y    = result * norm;
    t      = st->s1 << 9;
    st->s2 ^= st->s0;  st->s3 ^= st->s1;
    st->s1 ^= st->s2;  st->s0 ^= st->s3;
    st->s2 ^= t;       st->s3 = rotl32(st->s3, 11);

    // — tercera muestra —
    result = st->s0 + st->s3;
    r.z    = result * norm;
    t      = st->s1 << 9;
    st->s2 ^= st->s0;  st->s3 ^= st->s1;
    st->s1 ^= st->s2;  st->s0 ^= st->s3;
    st->s2 ^= t;       st->s3 = rotl32(st->s3, 11);

    // — cuarta muestra —
    result = st->s0 + st->s3;
    r.w    = result * norm;
    // estado ya modificado

    return r;
}

__global__ void photon_kernel_xoshiro(float* heats, float* heats_squared, unsigned int photons_per_thread) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // apuntadores en shared memory
    __shared__ float s_heats[SHELLS];
    __shared__ float s_heats_sq[SHELLS];

    // Inicialización de shared (por los primeros SHELLS hilos)
    if (tid < SHELLS) {
        s_heats[tid]    = 0.0f;
        s_heats_sq[tid] = 0.0f;
    }
    
    __syncthreads();

    Xoshiro128pState state;
    uint64_t seed = clock64() + (uint64_t)idx;
    xoshiro128p_init(seed, &state);

    // Precompute constants
    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4f / MICRONS_PER_SHELL / (MU_A + MU_S);

    // Photon state
    float x = 0.0f, y = 0.0f, z = 0.0f;
    float u = 0.0f, v = 0.0f, w = 1.0f;
    float weight = 1.0f;

    unsigned int remaining_photons = photons_per_thread;

    // Photon life loop
    while (remaining_photons) {
        float4 rnd4 = xoshiro128p_next4(&state);
        // Sample step length
        float rnd = rnd4.w;
        float t = -logf(rnd);
        x += t * u;
        y += t * v;
        z += t * w;

        // Determine shell
        int shell = (int)(sqrtf(x*x + y*y + z*z) * shells_per_mfp);
        if (shell >= SHELLS) shell = SHELLS - 1;

        // Deposit energy
        float deposit = (1.0f - albedo) * weight;
        atomicAdd(&s_heats[shell], deposit);
        atomicAdd(&s_heats_sq[shell], deposit*deposit);

        // Update weight
        weight *= albedo;

        // Roulette for low-weight photons
        if (weight < 0.001f) {
            if (rnd4.x > 0.1f) {
                // Photon is absorbed
                remaining_photons--;
                x = 0.0f;
                y = 0.0f;
                z = 0.0f;
                u = 0.0f;
                v = 0.0f;
                w = 1.0f;
                weight = 1.0f;
                continue;
            } else {
                weight *= 10.0f;
            }
        }

        float xi1, xi2, s, r, sin, cos;

        s = rnd4.y;
        r = sqrtf(s);
        sincospif(2.0f * rnd4.z, &sin, &cos);
        xi1 = r * cos;
        xi2 = r * sin;

        u = 2.0f * s - 1.0f;
        float factor = 2.0f * sqrtf(1.0f - s);
        v = xi1 * factor;
        w = xi2 * factor;
    }

    __syncthreads();

    if (tid < SHELLS) {
        atomicAdd(&heats[tid], s_heats[tid]);
        atomicAdd(&heats_squared[tid], s_heats_sq[tid]);
    }
}

__global__ void photon_kernel_rsqrt(float* heats, float* heats_squared, unsigned int photons_per_thread) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // apuntadores en shared memory
    __shared__ float s_heats[SHELLS];
    __shared__ float s_heats_sq[SHELLS];

    if (GPU_THREADS < SHELLS) {
        if (tid == 0) {
            for (int i = 0; i < SHELLS; i++) {
                s_heats[i]    = 0.0f;
                s_heats_sq[i] = 0.0f;
            }
        }
    } else {
        if (tid < SHELLS) {
            s_heats[tid]    = 0.0f;
            s_heats_sq[tid] = 0.0f;
        }
    }
    
    __syncthreads();

    Xoshiro128pState state;
    uint64_t seed = clock64() + (uint64_t)idx;
    xoshiro128p_init(seed, &state);

    // Precompute constants
    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4f / MICRONS_PER_SHELL / (MU_A + MU_S);

    // Photon state
    float x = 0.0f, y = 0.0f, z = 0.0f;
    float u = 0.0f, v = 0.0f, w = 1.0f;
    float weight = 1.0f;

    unsigned int remaining_photons = photons_per_thread;

    // Photon life loop
    while (remaining_photons) {
        float4 rnd4 = xoshiro128p_next4(&state);
        // Sample step length
        float rnd = rnd4.w;
        float t = -logf(rnd);
        x += t * u;
        y += t * v;
        z += t * w;

        // Determine shell
        float aux = x*x + y*y + z*z;
        float inv_r = rsqrtf(aux);
        int shell = min(int(inv_r * aux * shells_per_mfp), SHELLS-1);

        // Deposit energy
        float deposit = (1.0f - albedo) * weight;
        atomicAdd(&s_heats[shell], deposit);
        atomicAdd(&s_heats_sq[shell], deposit*deposit);

        // Update weight
        weight *= albedo;

        // Roulette for low-weight photons
        if (weight < 0.001f) {
            if (rnd4.x > 0.1f) {
                // Photon is absorbed
                remaining_photons--;
                x = 0.0f;
                y = 0.0f;
                z = 0.0f;
                u = 0.0f;
                v = 0.0f;
                w = 1.0f;
                weight = 1.0f;
                continue;
            } else {
                weight *= 10.0f;
            }
        }

        float xi1, xi2, s, r, sin, cos;

        s = rnd4.y;
        r = rsqrtf(s) * s;
        sincospif(2.0f * rnd4.z, &sin, &cos);
        xi1 = r * cos;
        xi2 = r * sin;

        u = 2.0f * s - 1.0f;
        float temp = 1.0f - s;
        float factor = 2.0f * rsqrtf(temp) * temp;
        v = xi1 * factor;
        w = xi2 * factor;
    }

    __syncthreads();

    if (tid < SHELLS) {
        atomicAdd(&heats[tid], s_heats[tid]);
        atomicAdd(&heats_squared[tid], s_heats_sq[tid]);
    }

    if (GPU_THREADS < SHELLS) {
        if (tid == 0) {
            for (int i = 0; i < SHELLS; i++) {
                atomicAdd(&heats[i], s_heats[i]);
                atomicAdd(&heats_squared[i], s_heats_sq[i]);
            }
        }
    } else {
        if (tid < SHELLS) {
            atomicAdd(&heats[tid], s_heats[tid]);
            atomicAdd(&heats_squared[tid], s_heats_sq[tid]);
        }
    }
}

void launch_simulation(float *h_heats, float *h_heats_sq, double *elapsed_time) {
    float *d_heats, *d_heats_sq;
    checkCudaCall(hipMalloc(&d_heats,  SHELLS * sizeof(float)));
    checkCudaCall(hipMalloc(&d_heats_sq, SHELLS * sizeof(float)));
    checkCudaCall(hipMemset(d_heats,  0, SHELLS * sizeof(float)));
    checkCudaCall(hipMemset(d_heats_sq, 0, SHELLS * sizeof(float)));

    hipEvent_t start, stop;
    checkCudaCall(hipEventCreate(&start));
    checkCudaCall(hipEventCreate(&stop));

    //int blocks = (PHOTONS / GPU_PHOTONS_PER_THREAD) / GPU_THREADS;
    unsigned int photons_per_thread = PHOTONS / (GPU_BLOCKS * GPU_THREADS);
    printf("# Launching %d blocks with %d threads each and %d photons per thread\n", GPU_BLOCKS, GPU_THREADS, photons_per_thread);

    checkCudaCall(hipEventRecord(start, 0));

    // kernel v1
    //photon_kernel_atomic_global<<<GPU_BLOCKS, GPU_THREADS>>>(d_heats, d_heats_sq, photons_per_thread);

    // kernel v2
    //photon_kernel_shared<<<GPU_BLOCKS, GPU_THREADS>>>(d_heats, d_heats_sq, photons_per_thread);

    // kernel v3
    //photon_kernel_polares<<<GPU_BLOCKS, GPU_THREADS>>>(d_heats, d_heats_sq, photons_per_thread);

    // kernel v7
    //photon_kernel_xoshiro<<<GPU_BLOCKS, GPU_THREADS>>>(d_heats, d_heats_sq, photons_per_thread);

    // kernel v8
    photon_kernel_rsqrt<<<GPU_BLOCKS, GPU_THREADS>>>(d_heats, d_heats_sq, photons_per_thread);

    checkCudaCall(hipEventRecord(stop, 0));
    checkCudaCall(hipEventSynchronize(stop));

    float ms = 0.0f;
    checkCudaCall(hipEventElapsedTime(&ms, start, stop));

    *elapsed_time = ms / 1000.0;

    checkCudaCall(hipMemcpy(h_heats, d_heats, SHELLS * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaCall(hipMemcpy(h_heats_sq, d_heats_sq, SHELLS * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipEventDestroy(start));
    checkCudaCall(hipEventDestroy(stop));
    checkCudaCall(hipFree(d_heats));
    checkCudaCall(hipFree(d_heats_sq));
}
