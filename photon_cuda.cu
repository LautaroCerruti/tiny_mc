#include "hip/hip_runtime.h"
#include "photon_cuda.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include <math.h>
#include "params.h"
#include "hip/hip_runtime_api.h"
#include <cstdint>

struct Xoshiro128pState {
    uint32_t s0, s1, s2, s3;
};

__device__ __forceinline__ uint32_t rotl32(uint32_t x, int k) {
    return (x << k) | (x >> (32 - k));
}

__device__ __forceinline__ uint64_t splitmix64(uint64_t &state) {
    uint64_t z = (state += 0x9E3779B97F4A7C15ULL);
    z = (z ^ (z >> 30)) * 0xBF58476D1CE4E5B9ULL;
    z = (z ^ (z >> 27)) * 0x94D049BB133111EBULL;
    return z ^ (z >> 31);
}

__device__ __forceinline__ void xoshiro128p_init(uint64_t seed, Xoshiro128pState *st) {
    uint64_t sm = seed;
    st->s0 = (uint32_t)splitmix64(sm);
    st->s1 = (uint32_t)splitmix64(sm);
    st->s2 = (uint32_t)splitmix64(sm);
    st->s3 = (uint32_t)splitmix64(sm);
}

__device__ __forceinline__ float4 xoshiro128p_next4(Xoshiro128pState *st) {
    const float norm = 2.3283064365386963e-10f; // 1/2^32

    float4 r;
    uint32_t result, t;

    result = st->s0 + st->s3;
    r.x    = result * norm;
    t      = st->s1 << 9;
    st->s2 ^= st->s0;  st->s3 ^= st->s1;
    st->s1 ^= st->s2;  st->s0 ^= st->s3;
    st->s2 ^= t;       st->s3 = rotl32(st->s3, 11);

    result = st->s0 + st->s3;
    r.y    = result * norm;
    t      = st->s1 << 9;
    st->s2 ^= st->s0;  st->s3 ^= st->s1;
    st->s1 ^= st->s2;  st->s0 ^= st->s3;
    st->s2 ^= t;       st->s3 = rotl32(st->s3, 11);

    result = st->s0 + st->s3;
    r.z    = result * norm;
    t      = st->s1 << 9;
    st->s2 ^= st->s0;  st->s3 ^= st->s1;
    st->s1 ^= st->s2;  st->s0 ^= st->s3;
    st->s2 ^= t;       st->s3 = rotl32(st->s3, 11);

    result = st->s0 + st->s3;
    r.w    = result * norm;

    return r;
}

__global__ void photon_kernel(float* heats, float* heats_squared, unsigned int photons_per_thread) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float s_heats[SHELLS];
    __shared__ float s_heats_sq[SHELLS];

    if (GPU_THREADS < SHELLS) {
        if (tid == 0) {
            for (int i = 0; i < SHELLS; i++) {
                s_heats[i]    = 0.0f;
                s_heats_sq[i] = 0.0f;
            }
        }
    } else {
        if (tid < SHELLS) {
            s_heats[tid]    = 0.0f;
            s_heats_sq[tid] = 0.0f;
        }
    }
    
    __syncthreads();

    Xoshiro128pState state;
    uint64_t seed = clock64() + (uint64_t)idx;
    xoshiro128p_init(seed, &state);

    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4f / MICRONS_PER_SHELL / (MU_A + MU_S);

    float x = 0.0f, y = 0.0f, z = 0.0f;
    float u = 0.0f, v = 0.0f, w = 1.0f;
    float weight = 1.0f;

    unsigned int remaining_photons = photons_per_thread;

    while (remaining_photons) {
        float4 rnd4 = xoshiro128p_next4(&state);
        float rnd = rnd4.w;
        float t = -logf(rnd);
        x += t * u;
        y += t * v;
        z += t * w;

        float aux = x*x + y*y + z*z;
        float inv_r = rsqrtf(aux);
        int shell = min(int(inv_r * aux * shells_per_mfp), SHELLS-1);

        float deposit = (1.0f - albedo) * weight;
        atomicAdd(&s_heats[shell], deposit);
        atomicAdd(&s_heats_sq[shell], deposit*deposit);

        weight *= albedo;

        if (weight < 0.001f) {
            if (rnd4.x > 0.1f) {
                remaining_photons--;
                x = 0.0f;
                y = 0.0f;
                z = 0.0f;
                u = 0.0f;
                v = 0.0f;
                w = 1.0f;
                weight = 1.0f;
                continue;
            } else {
                weight *= 10.0f;
            }
        }

        float xi1, xi2, s, r, sin, cos;

        s = rnd4.y;
        r = rsqrtf(s) * s;
        sincospif(2.0f * rnd4.z, &sin, &cos);
        xi1 = r * cos;
        xi2 = r * sin;

        u = 2.0f * s - 1.0f;
        float temp = 1.0f - s;
        float factor = 2.0f * rsqrtf(temp) * temp;
        v = xi1 * factor;
        w = xi2 * factor;
    }

    __syncthreads();

    if (tid < SHELLS) {
        atomicAdd(&heats[tid], s_heats[tid]);
        atomicAdd(&heats_squared[tid], s_heats_sq[tid]);
    }

    if (GPU_THREADS < SHELLS) {
        if (tid == 0) {
            for (int i = 0; i < SHELLS; i++) {
                atomicAdd(&heats[i], s_heats[i]);
                atomicAdd(&heats_squared[i], s_heats_sq[i]);
            }
        }
    } else {
        if (tid < SHELLS) {
            atomicAdd(&heats[tid], s_heats[tid]);
            atomicAdd(&heats_squared[tid], s_heats_sq[tid]);
        }
    }
}

void launch_simulation(float *h_heats, float *h_heats_sq, double *elapsed_time) {
    float *d_heats, *d_heats_sq;
    checkCudaCall(hipMalloc(&d_heats,  SHELLS * sizeof(float)));
    checkCudaCall(hipMalloc(&d_heats_sq, SHELLS * sizeof(float)));
    checkCudaCall(hipMemset(d_heats,  0, SHELLS * sizeof(float)));
    checkCudaCall(hipMemset(d_heats_sq, 0, SHELLS * sizeof(float)));

    hipEvent_t start, stop;
    checkCudaCall(hipEventCreate(&start));
    checkCudaCall(hipEventCreate(&stop));

    unsigned int photons_per_thread = PHOTONS / (GPU_BLOCKS * GPU_THREADS);
    printf("# Launching %d blocks with %d threads each and %d photons per thread\n", GPU_BLOCKS, GPU_THREADS, photons_per_thread);

    checkCudaCall(hipEventRecord(start, 0));

    photon_kernel<<<GPU_BLOCKS, GPU_THREADS>>>(d_heats, d_heats_sq, photons_per_thread);

    checkCudaCall(hipEventRecord(stop, 0));
    checkCudaCall(hipEventSynchronize(stop));

    float ms = 0.0f;
    checkCudaCall(hipEventElapsedTime(&ms, start, stop));

    *elapsed_time = ms / 1000.0;

    checkCudaCall(hipMemcpy(h_heats, d_heats, SHELLS * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaCall(hipMemcpy(h_heats_sq, d_heats_sq, SHELLS * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipEventDestroy(start));
    checkCudaCall(hipEventDestroy(stop));
    checkCudaCall(hipFree(d_heats));
    checkCudaCall(hipFree(d_heats_sq));
}
